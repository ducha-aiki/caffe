#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/custom_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ELUForward(const int n, const Dtype* in, Dtype* out,
    Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : (exp(in[index]) - Dtype(1)) * alpha;
  }
}
template <typename Dtype>
__global__ void ELUSymmmetricForward(const int n, const Dtype* in, Dtype* out,
    Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? (Dtype(1) - exp(-in[index])) * alpha : (exp(in[index]) - Dtype(1)) * alpha;
  }
}
template <typename Dtype>
void ELULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  
  Dtype alpha = this->layer_param_.elu_param().alpha();
  const bool symmetric_mode =  this->layer_param_.elu_param().symmetric_mode();
  if (symmetric_mode) {
   // NOLINT_NEXT_LINE(whitespace/operators)
  ELUSymmmetricForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, alpha);
  CUDA_POST_KERNEL_CHECK;
  }else{    
  // NOLINT_NEXT_LINE(whitespace/operators)
  ELUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, alpha);
  CUDA_POST_KERNEL_CHECK;
  }
}

template <typename Dtype>
__global__ void ELUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data,const Dtype* out_data, Dtype* out_diff, Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * (out_data[index] + alpha));
  }
}
template <typename Dtype>
__global__ void ELUSymmetricBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data,const Dtype* out_data, Dtype* out_diff, Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)* (-out_data[index] + alpha)
        + (in_data[index] <= 0) * (out_data[index] + alpha));
  }
}
template <typename Dtype>
void ELULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype alpha = this->layer_param_.elu_param().alpha();
    
      const bool symmetric_mode =  this->layer_param_.elu_param().symmetric_mode();
  if (symmetric_mode) {
          // NOLINT_NEXT_LINE(whitespace/operators)
    ELUSymmetricBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, top_data, bottom_diff, alpha);
    CUDA_POST_KERNEL_CHECK;
  } else {
    // NOLINT_NEXT_LINE(whitespace/operators)
    ELUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, top_data, bottom_diff, alpha);
    CUDA_POST_KERNEL_CHECK;
  }
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ELULayer);


}  // namespace caffe
