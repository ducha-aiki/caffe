#include "hip/hip_runtime.h"
#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

#include <iostream>

namespace caffe {

template <typename Dtype>
__global__ void CroppingForward(const int count, const Dtype* in, Dtype* out,
    const int num, const int channel, const int height_in, const int width_in,
    const int startx, const int starty, const int height_out, const int width_out) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < count) {
    int w = index % width_out;
    index /= width_out;
    int h = index % height_out;
    index /= height_out;
    int c = index % channel;
    index /= channel;
    out[((index * channel + c) * height_out + h) * width_out + w] =
        in[((index * channel + c) * height_in + h + starty) * width_in + w + startx];
  }
}

template <typename Dtype>
void CroppingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = top[0]->count();
  // First, set all data to be zero for the boundary pixels
  CroppingForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, NUM_, CHANNEL_, HEIGHT_IN_, WIDTH_IN_,
      STARTX_, STARTY_, HEIGHT_OUT_, WIDTH_OUT_);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void CroppingBackward(const int count, const Dtype* in, Dtype* out,
    const int num, const int channel, const int height_in, const int width_in,
    const int startx, const int starty, const int height_out, const int width_out) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < count) {
    int w = index % width_out;
    index /= width_out;
    int h = index % height_out;
    index /= height_out;
    int c = index % channel;
    index /= channel;
    out[((index * channel + c) * height_in + h + starty) * width_in + w + startx] =
        in[((index * channel + c) * height_out + h) * width_out + w];
  }
}

template <typename Dtype>
void CroppingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = top[0]->count();
    CUDA_CHECK(hipMemset(bottom_diff, 0, sizeof(Dtype) * (*bottom)[0]->count()));
    CroppingBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_diff, NUM_, CHANNEL_, HEIGHT_IN_, WIDTH_IN_,
        STARTX_, STARTY_, HEIGHT_OUT_, WIDTH_OUT_);
    CUDA_POST_KERNEL_CHECK;
  }
  return ;
}

}  // namespace caffe
